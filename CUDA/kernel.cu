#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <ctime>
#include <stdio.h>
#include "opencv2/opencv.hpp"
#include <cmath>
#include <algorithm>   
using namespace cv;
using namespace std;
#define CUDA_CALL(x) do { if((x) != hipSuccess) { \
    printf("Error %s (%d) at %s:%d\n", hipGetErrorString(x),x, __FILE__,__LINE__); \
    return EXIT_FAILURE;}} while(0)

// ���С����
#define BLOCKWIDTH 16
#define BLOCKHEIGHT 16

// ȫ�ֱ����洢����
Mat phi_;		//level set: ��
Mat src_;
Mat image_; //for showing��ʾͼ��
const int iterationnum_ = 30000;



// GPU�����洢����
//__constant__ int d_iterationnum = 500;
__constant__ float d_lambda1 = 1.0f;
__constant__ float d_lambda2 = 1.0f;
__constant__ float d_mu = 0.1 * 255 * 255;	//������
__constant__ float d_nu = 0.0;  //�����
//__constant__ float d_timestep = 1; //��t �촽����
__constant__ float d_timestep = 5; //��t �ɻ�����
//__constant__ float d_timestep = 5; //��t ���ܲ���
__constant__ float d_epsilon = 1.0;
__constant__ float d_k1 = 1 / CV_PI; // d_epsilon=1�����
__constant__ float d_k2 = 1.0; // d_epsilon=1�����
__constant__ float d_k3 = 2 / CV_PI; // d_epsilon=1�����
__constant__ float d_forntpro = 0.1; //ǰ����ֵ����ϵ��

// �ɻ�2���ž��뺯����ʼ������
//__constant__ float d_phi0centerx = 290.0; //��ʼ���ž��뺯������x
//__constant__ float d_phi0centery = 400.0; //��ʼ���ž��뺯������y
//__constant__ float d_phi0r = 450.0; //��ʼ���ž��뺯���뾶

// �ɻ�1���ž��뺯����ʼ������
__constant__ float d_phi0centerx = 160.0; //��ʼ���ž��뺯������x
__constant__ float d_phi0centery = 285.0; //��ʼ���ž��뺯������y
__constant__ float d_phi0r = 300.0; //��ʼ���ž��뺯���뾶

// �촽���ž��뺯����ʼ������
//__constant__ float d_phi0centerx = 400.0; //��ʼ���ž��뺯������x
//__constant__ float d_phi0centery = 240.0; //��ʼ���ž��뺯������y
//__constant__ float d_phi0r = 200.0; //��ʼ���ž��뺯���뾶

// ���ܷ��ž��뺯����ʼ������2
//__constant__ float d_phi0centerx = 250.0; //��ʼ���ž��뺯������x
//__constant__ float d_phi0centery = 250.0; //��ʼ���ž��뺯������y
//__constant__ float d_phi0r = 353.0; //��ʼ���ž��뺯���뾶


// GPUȫ�ֱ����洢����
__device__ float d_c1;
__device__ float d_c2;
__device__ float h1result;
__device__ float h2result;
__device__ float sum1result;
__device__ float sum2result;



// ����x������y�����ݶ�
__device__	void gradient(float* d_src, float* gradx, float* grady, int idx, int idy, uint imgWidth, uint imgHeight)
{
	float xtemp = 0, ytemp = 0;
 
	// ˮƽ�����ݶ�
	if (idx == 0)
	{
		xtemp = d_src[idy * imgWidth + (idx + 1)] - d_src[idy * imgWidth + idx];
	}
	else if (idx == imgWidth - 1)
	{
		xtemp = d_src[idy * imgWidth + idx] - d_src[idy * imgWidth + idx - 1];
	}
	else
	{
		xtemp = (d_src[idy * imgWidth + (idx + 1)] - d_src[idy * imgWidth + idx - 1]) / 2.0;
	}

	if (xtemp == 0)
	{
		gradx[idy * imgWidth + idx] = 1e-8;
	}
	else
	{
		gradx[idy * imgWidth + idx] = xtemp;
	}


	// ��ֱ�����ݶ�
	if (idy == 0)
	{
		ytemp = d_src[(idy + 1) * imgWidth + idx] - d_src[idy * imgWidth + idx];
	}
	else if (idy == imgHeight - 1)
	{
		ytemp = d_src[idy * imgWidth + idx] - d_src[(idy - 1) * imgWidth + idx];
	}
	else
	{
		ytemp = (d_src[(idy + 1) * imgWidth + idx] - d_src[(idy - 1) * imgWidth + idx]) / 2.0;
	}

	if (ytemp == 0)
	{
		grady[idy * imgWidth + idx] = 1e-8;
	}
	else
	{
		grady[idy * imgWidth + idx] = ytemp;
	}
}

// ����x�����ݶ�
__device__	void gradientx(float* d_src, float* gradx, int idx, int idy, uint imgWidth, uint imgHeight)
{
	float xtemp = 0;

	// ˮƽ�����ݶ�
	if (idx == 0)
	{
		xtemp = d_src[idy * imgWidth + (idx + 1)] - d_src[idy * imgWidth + idx];
	}
	else if (idx == imgWidth - 1)
	{
		xtemp = d_src[idy * imgWidth + idx] - d_src[idy * imgWidth + idx - 1];
	}
	else
	{
		xtemp = (d_src[idy * imgWidth + (idx + 1)] - d_src[idy * imgWidth + idx - 1]) / 2.0;
	}

	if (xtemp == 0)
	{
		gradx[idy * imgWidth + idx] = 1e-8;
	}
	else
	{
		gradx[idy * imgWidth + idx] = xtemp;
	}
}


// ����y�����ݶ�
__device__	void gradienty(float* d_src, float* grady, int idx, int idy, uint imgWidth, uint imgHeight)
{
	float ytemp = 0;

	// ��ֱ�����ݶ�
	if (idy == 0)
	{
		ytemp = d_src[(idy + 1) * imgWidth + idx] - d_src[idy * imgWidth + idx];
	}
	else if (idy == imgHeight - 1)
	{
		ytemp = d_src[idy * imgWidth + idx] - d_src[(idy - 1) * imgWidth + idx];
	}
	else
	{
		ytemp = (d_src[(idy + 1) * imgWidth + idx] - d_src[(idy - 1) * imgWidth + idx]) / 2.0;
	}

	if (ytemp == 0)
	{
		grady[idy * imgWidth + idx] = 1e-8;
	}
	else
	{
		grady[idy * imgWidth + idx] = ytemp;
	}
}


// zong��ά�鲢����
__device__ void gpuReduction2zong(float* d_a, float* d_value, uint imgWidth, uint imgHeight)
{
	__shared__ float temph1[BLOCKHEIGHT][BLOCKWIDTH], temph2[BLOCKHEIGHT][BLOCKWIDTH], tempsum1[BLOCKHEIGHT][BLOCKWIDTH], tempsum2[BLOCKHEIGHT][BLOCKWIDTH];
	// idx Ϊ�кţ�������
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	// idy Ϊ�кţ�������
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	if (idx < imgWidth && idy < imgHeight)
	{
		int ycrow = imgHeight % BLOCKHEIGHT;
		int yccol = imgWidth % BLOCKWIDTH;
		// ����Լ������ȡ
		float h1per = d_a[idy * imgWidth + idx];
		//printf("%f\n", h1per);
		float h2per = 1 - d_a[idy * imgWidth + idx];
		float sum1per = d_a[idy * imgWidth + idx] * d_value[idy * imgWidth + idx];
		float sum2per = (1 - d_a[idy * imgWidth + idx]) * d_value[idy * imgWidth + idx];

		// �����ڿ鳬��ͼƬ�����
		if (ycrow > 0 || yccol > 0)
		{
			int ycblocky = imgHeight / BLOCKHEIGHT;
			int ycblockx = imgWidth / BLOCKWIDTH;

			if ((blockIdx.x > (ycblockx - 1)) || (blockIdx.y > (ycblocky - 1)))
			{
				atomicAdd(&h1result, h1per);
				atomicAdd(&h2result, h2per);
				atomicAdd(&sum1result, sum1per);
				atomicAdd(&sum2result, sum2per);

				//printf("%d,%d\n", blockIdx.x, blockIdx.y);
			}
			else
			{
				temph1[threadIdx.y][threadIdx.x] = h1per;
				temph2[threadIdx.y][threadIdx.x] = h2per;
				tempsum1[threadIdx.y][threadIdx.x] = sum1per;
				tempsum2[threadIdx.y][threadIdx.x] = sum2per;

				__syncthreads();

				// ���н��й�Լ
				for (int i = (BLOCKWIDTH >> 1); i > 0; i >>= 1) {
					if (threadIdx.x < i) {
						temph1[threadIdx.y][threadIdx.x] += temph1[threadIdx.y][threadIdx.x + i];
						temph2[threadIdx.y][threadIdx.x] += temph2[threadIdx.y][threadIdx.x + i];
						tempsum1[threadIdx.y][threadIdx.x] += tempsum1[threadIdx.y][threadIdx.x + i];
						tempsum2[threadIdx.y][threadIdx.x] += tempsum2[threadIdx.y][threadIdx.x + i];
					}
					__syncthreads();
				}
				//__syncthreads();
				//printf("%f\n", temp[k][0]);
				// ���н��й�Լ
				if (threadIdx.x == 0)
				{
					for (int i = (BLOCKHEIGHT >> 1); i > 0; i >>= 1) {
						if (threadIdx.y < i) {
							temph1[threadIdx.y][threadIdx.x] += temph1[threadIdx.y + i][threadIdx.x];
							temph2[threadIdx.y][threadIdx.x] += temph2[threadIdx.y + i][threadIdx.x];
							tempsum1[threadIdx.y][threadIdx.x] += tempsum1[threadIdx.y + i][threadIdx.x];
							tempsum2[threadIdx.y][threadIdx.x] += tempsum2[threadIdx.y + i][threadIdx.x];
						}
						__syncthreads();
					}
				}
				__syncthreads();
				// ȫ�ֹ�Լ�������
				if (threadIdx.x == 0 && threadIdx.y == 0)
				{
					atomicAdd(&h1result, temph1[0][0]);
					atomicAdd(&h2result, temph2[0][0]);
					atomicAdd(&sum1result, tempsum1[0][0]);
					atomicAdd(&sum2result, tempsum2[0][0]);
				}
			}
		}
		// �������ڿ鳬��ͼƬ�����
		else
		{
			temph1[threadIdx.y][threadIdx.x] = h1per;
			temph2[threadIdx.y][threadIdx.x] = h2per;
			tempsum1[threadIdx.y][threadIdx.x] = sum1per;
			tempsum2[threadIdx.y][threadIdx.x] = sum2per;

			__syncthreads();

			// ���н��й�Լ
			for (int i = (BLOCKWIDTH >> 1); i > 0; i >>= 1) {
				if (threadIdx.x < i) {
					temph1[threadIdx.y][threadIdx.x] += temph1[threadIdx.y][threadIdx.x + i];
					temph2[threadIdx.y][threadIdx.x] += temph2[threadIdx.y][threadIdx.x + i];
					tempsum1[threadIdx.y][threadIdx.x] += tempsum1[threadIdx.y][threadIdx.x + i];
					tempsum2[threadIdx.y][threadIdx.x] += tempsum2[threadIdx.y][threadIdx.x + i];
				}
				__syncthreads();
			}
			//__syncthreads();
			//printf("%f\n", temp[k][0]);
			// ���н��й�Լ
			if (threadIdx.x == 0)
			{
				for (int i = (BLOCKHEIGHT >> 1); i > 0; i >>= 1) {
					if (threadIdx.y < i) {
						temph1[threadIdx.y][threadIdx.x] += temph1[threadIdx.y + i][threadIdx.x];
						temph2[threadIdx.y][threadIdx.x] += temph2[threadIdx.y + i][threadIdx.x];
						tempsum1[threadIdx.y][threadIdx.x] += tempsum1[threadIdx.y + i][threadIdx.x];
						tempsum2[threadIdx.y][threadIdx.x] += tempsum2[threadIdx.y + i][threadIdx.x];
					}
					__syncthreads();
				}
				//printf("%f\n", temp[0][0]);
			}
			__syncthreads();
			// ȫ�ֹ�Լ�������
			if (threadIdx.x == 0 && threadIdx.y == 0)
			{
				atomicAdd(&h1result, temph1[0][0]);
				atomicAdd(&h2result, temph2[0][0]);
				atomicAdd(&sum1result, tempsum1[0][0]);
				atomicAdd(&sum2result, tempsum2[0][0]);
			}
		}
	}
}


__global__ void evolvingArg(float* d_phi, float* d_dirac, float* d_heaviside, float* d_curv, float* d_src, float* d_dx, float* d_dy, float* d_dxx,
	float* d_dyy, uint imgWidth, uint imgHeight) {

	// idx Ϊ�кţ�������
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	// idy Ϊ�кţ�������
	int idy = threadIdx.y + blockIdx.y * blockDim.y;

	if (idx < imgWidth && idy < imgHeight)
	{
		d_dirac[idy * imgWidth + idx] = d_k1 / (d_k2 + powf(d_phi[idy * imgWidth + idx], 2));
		d_heaviside[idy * imgWidth + idx] = 0.5 * (1.0 + d_k3 * atanf(d_phi[idy * imgWidth + idx] / d_epsilon));
		// ���ʼ���
		gradient(d_src, d_dx, d_dy, idx, idy, imgWidth, imgHeight);
		float norm = powf(d_dx[idy * imgWidth + idx] * d_dx[idy * imgWidth + idx] + d_dy[idy * imgWidth + idx] * d_dy[idy * imgWidth + idx], 0.5);
		d_dx[idy * imgWidth + idx] = d_dx[idy * imgWidth + idx] / norm;
		d_dy[idy * imgWidth + idx] = d_dy[idy * imgWidth + idx] / norm;
	}
	else
	{
		return;
	}

}



__global__ void evolvingCrvAndAvg(float* d_phi, float* d_dirac, float* d_heaviside, float* d_curv, float* d_src, float* d_dx, float* d_dy, float* d_dxx,
	float* d_dyy, uint imgWidth, uint imgHeight) {
	// idx Ϊ�кţ�������
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	// idy Ϊ�кţ�������
	int idy = threadIdx.y + blockIdx.y * blockDim.y;

	if (idx < imgWidth && idy < imgHeight)
	{
		gradientx(d_dx, d_dxx, idx, idy, imgWidth, imgHeight);
		gradienty(d_dy, d_dyy, idx, idy, imgWidth, imgHeight);
		d_curv[idy * imgWidth + idx] = d_dxx[idy * imgWidth + idx] + d_dyy[idy * imgWidth + idx];

		float value = d_src[idy * imgWidth + idx];
		float h = d_heaviside[idy * imgWidth + idx];
		if (idx == 0 && idy == 0)
		{
			h1result = 0;
			h2result = 0;
			sum1result = 0;
			sum2result = 0;
		}
		__threadfence();
		gpuReduction2zong(d_heaviside, d_src, imgWidth, imgHeight);

	}
	else
	{
		return;
	}

}
__global__ void evolvingProAndCheck(float* d_phi, float* d_dirac, float* d_heaviside, float* d_curv, float* d_src, float* d_dx, float* d_dy, float* d_dxx,
	float* d_dyy, int* d_flag, uint imgWidth, uint imgHeight) {
	// idx Ϊ�кţ�������
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	// idy Ϊ�кţ�������
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	//float c1, c2;
	if (idx == 0 && idy == 0)
	{
		d_c1 = d_forntpro * sum1result / (h1result + 1e-10);
		d_c2 = sum2result / (h2result + 1e-10);
		*d_flag = 0;
		//printf("h1:%e,h2:%e,sum1:%e,sum2:%e,c1:%e,c2:%e\n", h1result,h2result,sum1result,sum2result, c1, c2);
	}
	__threadfence();
	if (idx < imgWidth && idy < imgHeight)
	{
		float curv = d_curv[idy * imgWidth + idx];
		float dirac = d_dirac[idy * imgWidth + idx];
		float u0 = d_src[idy * imgWidth + idx];

		float lengthTerm = d_mu * dirac * curv;
		float areamterm = d_nu * dirac;
		float fittingterm = dirac * (-d_lambda1 * powf(u0 - d_c1, 2) + d_lambda2 * powf(u0 - d_c2, 2));
		float term = lengthTerm + areamterm + fittingterm;
		float newphi = d_phi[idy * imgWidth + idx] + d_timestep * term;
		float oldphi = d_phi[idy * imgWidth + idx];
		d_phi[idy * imgWidth + idx] = newphi;
		if (*d_flag == 0)
		{
			if (oldphi * newphi < 0)
			{
				*d_flag = 1;
			}
			__threadfence();
		}
	}
	else
	{
		return;
	}

}

__global__ void cudaInitializePhi(float* d_phi, uint imgWidth, uint imgHeight) {
	// idx Ϊ�кţ�������
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	// idy Ϊ�кţ�������
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	if (idx < imgWidth && idy < imgHeight)
	{
		float value = -sqrtf(powf((idx - d_phi0centerx), 2) + powf((idy - d_phi0centery), 2)) + d_phi0r;
		if (abs(value) < 1e-3)
		{
			//����ˮƽ��������
			d_phi[idy * imgWidth + idx] = 0;
		}
		else
		{
			// ����ˮƽ���ڣ�Ϊ��
			// ����ˮƽ���⣺Ϊ��
			d_phi[idy * imgWidth + idx] = value;
		}
	}
	else
	{
		return;
	}

}


// �����ݻ�����
void showEvolving()
{
	Mat image = image_.clone();
	Mat mask = phi_ >= 0;
	// ���п�����
	cv::dilate(mask, mask, Mat(), Point(-1, -1), 3);
	cv::erode(mask, mask, Mat(), Point(-1, -1), 3);
	vector<vector<Point>> contours;
	findContours(mask, contours, RETR_EXTERNAL, CHAIN_APPROX_NONE);
	drawContours(image, contours, -1, CV_RGB(0, 255, 0), 2);
	//namedWindow("Evolving");
	//moveWindow("Evolving", 600, 100);
	imshow("Evolving", image);
	waitKey(1);
}

// ����ˮƽ���ݻ�ͼ
void showLevelsetEvolving()
{
	Mat phic = phi_.clone();
	Mat mask = phi_ >= 0;
	Mat phinom;
	Mat phinomu;
	Mat phiev; // ˮƽ�������ݻ�չʾͼ��
	normalize(phic, phinom, 0, 255, NORM_MINMAX, -2, mask);
	phinom.convertTo(phinomu, CV_8UC1);
	applyColorMap(phinomu, phiev, COLORMAP_JET);
	//namedWindow("EvolvingLevelset");
	//moveWindow("EvolvingLevelset", 122, 100);
	imshow("EvolvingLevelset", phiev);
	waitKey(1);
}

int main()
{
	Mat src;
	//src = imread("C:/Users/Neal Wang/Desktop/mouth.jpg");
	src = imread("C:/Users/Neal Wang/Desktop/plane.jpg");
	//src = imread("C:/Users/Neal Wang/Desktop/plane2.jpg");

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);

	if (src.channels() == 3)
	{
		cv::cvtColor(src, src_, COLOR_BGR2GRAY);
		src.copyTo(image_);
	}
	else if (src.channels() == 1)
	{
		src.copyTo(src_);
		cv::cvtColor(src, image_, COLOR_GRAY2BGR);
	}
	else
	{
		printf("���������ͨ����ͼƬ");
	}
	// ������ʼ��
	src_.convertTo(src_, CV_32FC1);
	phi_ = Mat::zeros(src_.size(), CV_32FC1);

	int imgHeight = src_.rows;
	int imgWidth = src_.cols;

	// ȫ�ֲ�������
	float* d_phi;
	float* d_dirac;
	float* d_heaviside;
	float* d_curv;
	float* d_src;
	// ���ʼ�����ز���
	float* d_dx, * d_dy, * d_dxx, * d_dyy;
	// �Ƿ�������ʶ����
	int* d_flag;

	CUDA_CALL(hipMalloc((void**)&d_phi, imgHeight * imgWidth * sizeof(float)));
	CUDA_CALL(hipMalloc((void**)&d_dirac, imgHeight * imgWidth * sizeof(float)));
	CUDA_CALL(hipMalloc((void**)&d_heaviside, imgHeight * imgWidth * sizeof(float)));
	CUDA_CALL(hipMalloc((void**)&d_curv, imgHeight * imgWidth * sizeof(float)));
	CUDA_CALL(hipMalloc((void**)&d_src, imgHeight * imgWidth * sizeof(float)));
	CUDA_CALL(hipMalloc((void**)&d_dx, imgHeight * imgWidth * sizeof(float)));
	CUDA_CALL(hipMalloc((void**)&d_dy, imgHeight * imgWidth * sizeof(float)));
	CUDA_CALL(hipMalloc((void**)&d_dxx, imgHeight * imgWidth * sizeof(float)));
	CUDA_CALL(hipMalloc((void**)&d_dyy, imgHeight * imgWidth * sizeof(float)));
	CUDA_CALL(hipMalloc((void**)&d_flag, sizeof(int)));

	// ���ݴ���
	CUDA_CALL(hipMemcpy(d_src, src_.data, imgHeight * imgWidth * sizeof(float), hipMemcpyHostToDevice));

	// block��grid����
	dim3 block1(BLOCKWIDTH, BLOCKHEIGHT);
	dim3 grid1((imgWidth + block1.x - 1) / block1.x, (imgHeight + block1.y - 1) / block1.y);

	int flag = 1;
	int convnum = iterationnum_;
	// ��ʼ�����ž��뺯��
	cudaInitializePhi << <grid1, block1 >> > (d_phi, imgWidth, imgHeight);
	// ����ָ��ݻ�����
	for (int i = 0; i < iterationnum_; i++)
	{
		// ������������
		evolvingArg << <grid1, block1 >> > (d_phi, d_dirac, d_heaviside, d_curv, d_src, d_dx, d_dy, d_dxx, d_dyy, imgWidth, imgHeight);

		evolvingCrvAndAvg << <grid1, block1 >> > (d_phi, d_dirac, d_heaviside, d_curv, d_src, d_dx, d_dy, d_dxx, d_dyy, imgWidth, imgHeight);

		evolvingProAndCheck << <grid1, block1 >> > (d_phi, d_dirac, d_heaviside, d_curv, d_src, d_dx, d_dy, d_dxx, d_dyy, d_flag, imgWidth, imgHeight);

		CUDA_CALL(hipMemcpy(&flag, d_flag, sizeof(int), hipMemcpyDeviceToHost));
		if (flag == 0)
		{
			convnum = i;
			break;
		}
		//CUDA_CALL(hipMemcpy(phi_.data, d_phi, imgHeight * imgWidth * sizeof(float), hipMemcpyDeviceToHost));
		//showEvolving();
		//showLevelsetEvolving();
	}
	CUDA_CALL(hipMemcpy(phi_.data, d_phi, imgHeight * imgWidth * sizeof(float), hipMemcpyDeviceToHost));
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	float elapsedTime;
	hipEventElapsedTime(&elapsedTime, start, stop);
	printf("����ʱ�䣺%3.1f ms\n", elapsedTime);
	printf("����������%d", convnum);

	showEvolving();
	showLevelsetEvolving();
	// �ͷ�Ԥ�ȷ�����Դ�
	hipFree(d_phi);
	hipFree(d_dirac);
	hipFree(d_heaviside);
	hipFree(d_curv);
	hipFree(d_src);
	hipFree(d_dx);
	hipFree(d_dy);
	hipFree(d_dxx);
	hipFree(d_dyy);
	hipFree(d_flag);



	waitKey(0);
	return 0;
}
